#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mm_math.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

MM::mat<double> gpu_mm(const MM::mat<double>& left, const MM::mat<double>& right);

__global__ void gpu_mm_kernel(int M, int N, int w, const double* d_left, const double* d_right, double* d_output)
{
	//Get the global indexes of the left and right arrays
	int lIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int rIdx = blockIdx.y * blockDim.y + threadIdx.y;

	//Make sure the thread is not trying to access out-of-bounds memory
	if (rIdx < N && lIdx < M)
	{
		double value = 0;
		for (int i = 0; i < w; i++)
		{
			value += d_left[lIdx * w + i] * d_right[rIdx + N * i];
		}
		d_output[lIdx * N + rIdx] = value;
	}

}

MM::mat<double> gpu_mm(const MM::mat<double>& left, const MM::mat<double>& right)
{

	if (left.columns() != right.rows())
	{
		printf("matrix dimension error\nLeft(column row): %i %i \nRight(column row): %i %i\n", left.columns(), left.rows(), right.columns(), right.rows());
	}
	else
	{
		int leftArraySize = left.columns() * left.rows();
		int rightArraySize = right.columns() * right.rows();
		int outputArraySize = left.rows() * right.columns();
		int n = left.columns() * left.rows() * right.columns();

		//Declare the arrays and allocate memory for them on the device
		double* d_left;
		double* d_right;
		double* d_output;
		gpuErrchk(hipMalloc(&d_left, leftArraySize * sizeof(double)));
		gpuErrchk(hipMalloc(&d_right, rightArraySize * sizeof(double)));
		gpuErrchk(hipMalloc(&d_output, outputArraySize * sizeof(double)));

		//Copy the values of the matrix object to a c-style array
		double* leftCArray = left.getCArray();
		double* rightCArray = right.getCArray();
		//Create an output matrix full of zeroes
		double* outputCArray = (double*)calloc(outputArraySize, sizeof(double));

		/*for(int i = 0;i<leftArraySize;i++)
		{
			printf("%f ", leftCArray[i]);
		}
		printf("\n");
		for(int i = 0;i<rightArraySize;i++)
		{
			printf("%f ", rightCArray[i]);
		}
		printf("\n");*/

		gpuErrchk(hipMemcpy(d_left, leftCArray, leftArraySize * sizeof(double), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_right, rightCArray, rightArraySize * sizeof(double), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_output, outputCArray, outputArraySize * sizeof(double), hipMemcpyHostToDevice));


		//Free the memory from the c-style arrays
		delete[] leftCArray;
		delete[] rightCArray;


		//Set the kernel launch parameters
		dim3 GRID((int)ceil((double)left.rows() / 32), (int)ceil((double)right.columns() / 32));
		dim3 BLOCK(32, 32);

		//Launch the kernel
		gpu_mm_kernel << <GRID, BLOCK >> > (left.rows(), right.columns(), left.columns(), d_left, d_right, d_output);
		gpuErrchk(hipPeekAtLastError());

		//Wait for the device to complete its work
		gpuErrchk(hipDeviceSynchronize());
		//Move the output array from the device to the host
		gpuErrchk(hipMemcpy(outputCArray, d_output, right.columns() * left.rows() * sizeof(double), hipMemcpyDeviceToHost));

		/*for(int i = 0;i<outputArraySize;i++)
		{
			printf("%f ", outputCArray[i]);
		}
		printf("\n");*/
		// Construct a matrix object based on the output c-style array
		MM::mat<double> outputmatrix(outputCArray, outputArraySize, left.rows(), right.columns());
		//printf("%s\n", outputmatrix.toStringFlipped(0).c_str());
		//Free the memory allocated on the gpu
		hipFree(d_left);
		hipFree(d_right);
		hipFree(d_output);
		delete[] outputCArray;
		return outputmatrix;
	}
	//The declaration promises to return something, so we return the left matrix in the case of error
	return left;
}
