#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mm_math.h"
#define N 32

MM::mat<double> gpu_mm(const MM::mat<double> &left, const MM::mat<double> &right);

__global__ void gpu_mm_kernel(int l_size, int l_y, double* d_left, int r_size, int r_y, double* d_right, double* d_output)
{
	//Get the global indexes of the left and right arrays
	int rIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int lIdx = blockIdx.y * blockDim.y + threadIdx.y;
	//Make sure the thread is not trying to access out-of-bounds memory
	if(rIdx < r_size && lIdx < l_size)
	{
		//Calculate which cell of the output array is incremented
		int outputIdx = rIdx/r_y + lIdx % l_y;

		//Multiply the elements and increase the matching element in the output array
		d_output[outputIdx] += d_left[lIdx] * d_right[rIdx];
	}
}

MM::mat<double> gpu_mm(const MM::mat<double> &left, const MM::mat<double> &right)
{
	
	if(left.columns() != right.rows())
	{
		printf("matrix dimension error\nLeft(column row): %i %i \nRight(column row): %i %i\n", left.columns(), left.rows(), right.columns(), right.rows());
	}
	else
	{
		int leftArraySize = left.columns() * left.rows();
		int rightArraySize = right.columns() * right.rows();
		int outputArraySize = left.rows() * right.columns();
		int n = left.columns() * left.rows() * right.columns();

		//Declare the arrays and allocate memory for them on the device
		double* d_left;
		double* d_right;
		double* d_output;
		hipMalloc(&d_left, leftArraySize * sizeof(double));
		hipMalloc(&d_right, rightArraySize * sizeof(double));
		hipMalloc(&d_output, outputArraySize * sizeof(double));

		//Copy the values of the matrix object to a c-style array
		double* leftCArray = left.getCArray();
		double* rightCArray = right.getCArray();
		hipMemcpy(d_left, leftCArray, left.columns() * left.rows() * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_right, rightCArray, right.columns() * right.rows() * sizeof(double), hipMemcpyHostToDevice);

		//Free the memory from the c-style arrays
		delete[] leftCArray;
		delete[] rightCArray;
		//Set the kernel launch parameters
		dim3 GRID(ceil(right.columns()/32*16), ceil(left.rows()/32*16));
		dim3 BLOCK(32*16, 32*16);
		
		//Launch the kernel
		gpu_mm_kernel<<<GRID, BLOCK>>>(leftArraySize, left.rows(), d_left, rightArraySize, right.rows(), d_right, d_output);
		//Move the output array from the device to the host
		double* outputCArray = (double*)malloc(right.columns() * left.rows() * sizeof(double*));
		hipMemcpy(outputCArray, d_output, right.columns() * left.rows() * sizeof(double), hipMemcpyDeviceToHost);
		// Construct a matrix object based on the output c-style array
		MM::mat<double> outputmatrix(outputCArray, outputArraySize, left.rows(), right.columns());
		//Free the memory allocated on the gpu
		hipFree(d_left);
		hipFree(d_right);
		hipFree(d_output);
		delete[] outputCArray;
		return outputmatrix;
	}
	//The declaration promises to return something, so we return the left matrix
	return left;
}

